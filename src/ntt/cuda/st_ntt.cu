#include "hip/hip_runtime.h"
#include "st_ntt.h"

#include "ntt/cuda/implementations/st_ntt_radix2.cuh"

#include <hip/hip_runtime_api.h>

#include <map>

using nttkernel = void (*)(int *vec, int mod);

std::map<uint, nttkernel> radix2{
    {2, stNttRadix2<2>},
    {4, stNttRadix2<4>},
    {8, stNttRadix2<8>},
    {16, stNttRadix2<16>},
    {32, stNttRadix2<32>},
    {64, stNttRadix2<64>},
    // {128, stNttRadix2<128, 7>},
};

float stNtt(std::span<int> vec, int size, int root, int mod, int batches, Radix radix) {
    int memsize = vec.size() * sizeof(int);
    int *vecGPU;

    float gpuTime;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    CCErr(hipMalloc(&vecGPU, memsize));
    CCErr(hipMemcpy(vecGPU, vec.data(), memsize, hipMemcpyHostToDevice));

    int *host_twiddles;
    CCErr(hipHostMalloc(&host_twiddles, size * sizeof(int)));
    int temp = 1;
    for (size_t i = 0; i < size / 2; ++i) {
        host_twiddles[i] = temp;
        temp = (temp * root) % mod;
    }

    CCErr(hipMemcpyToSymbol(HIP_SYMBOL(const_twiddles), host_twiddles, size / 2 * sizeof(int)));

    // Higher blockSize, bigger bottleneck on register usage per SM
    constexpr int blockSize = 512;
    int n = size;
    int n2 = n >> 1;
    int n4 = n >> 2;
    int n8 = n >> 3;
    int b2 = batches >> 1;
    int lN = std::bit_width(static_cast<uint>(n)) - 1;
    dim3 dimBlock;
    dim3 dimGrid;
    nttkernel kernel;

    // radix2
    kernel = radix2.at(n);
    dimBlock = dim3(n2, std::min(blockSize / n2, batches));
    dimGrid = dim3((n2 * batches + blockSize - 1) / blockSize);

    hipProfilerStart();
    hipEventRecord(start);
    kernel<<<dimGrid, dimBlock>>>(vecGPU, mod);
    CCErr(hipGetLastError());
    hipEventRecord(end);

    hipProfilerStop();
    hipEventSynchronize(end);
    hipEventElapsedTime(&gpuTime, start, end);

    CCErr(hipDeviceSynchronize());

    CCErr(hipMemcpy(vec.data(), vecGPU, memsize, hipMemcpyDeviceToHost));
    CCErr(hipFree(vecGPU));
    CCErr(hipHostFree(host_twiddles));

    return gpuTime;
}